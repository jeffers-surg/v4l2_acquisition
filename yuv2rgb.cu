#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2015, NVIDIA CORPORATION. All rights reserved.
 *
 * Permission is hereby granted, free of charge, to any person obtaining a
 * copy of this software and associated documentation files (the "Software"),
 * to deal in the Software without restriction, including without limitation
 * the rights to use, copy, modify, merge, publish, distribute, sublicense,
 * and/or sell copies of the Software, and to permit persons to whom the
 * Software is furnished to do so, subject to the following conditions:
 *
 * The above copyright notice and this permission notice shall be included in
 * all copies or substantial portions of the Software.
 *
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
 * IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
 * FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT.  IN NO EVENT SHALL
 * THE AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
 * LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING
 * FROM, OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER
 * DEALINGS IN THE SOFTWARE.
 */

#include <hip/hip_runtime.h>
#include "yuv2rgb.cuh"

__device__ inline float clamp(float val, float mn, float mx)
{
	return (val >= mn)? ((val <= mx)? val : mx) : mn;
}

/**
 * @brief Retrieve the index for the current process thread. Useful for indexing into images.
 * 
 * @return __device__ 
 */
__device__ int getGlobalIdx_2D_2D(){
int blockId = blockIdx.x + blockIdx.y * gridDim.x;
int threadId = blockId * (blockDim.x * blockDim.y)
 + (threadIdx.y * blockDim.x) + threadIdx.x;
return threadId;
}

#if 0
__global__ void gpuConvertYUYVtoRGB_kernel(unsigned char *src, unsigned char *dst,
		unsigned int width, unsigned int height)
{
	int index = getGlobalIdx_2D_2D();
    int x = index % (2 * 1536);
    int y = index / (2 * 1536);

	int params_width = 1536;
	int params_bpp = 3;

	bool white = false; //y % 100 < 5 || y %100 > 95;
    dst[x * params_bpp + y * params_width*2 * params_bpp + 0] = src[x * params_bpp + y * params_width*2 * params_bpp + 0];
	dst[x * params_bpp + y * params_width*2 * params_bpp + 1] = src[x * params_bpp + y * params_width*2 * params_bpp + 1];
	dst[x * params_bpp + y * params_width*2 * params_bpp + 2] = src[x * params_bpp + y * params_width*2 * params_bpp + 2];


    // dst[x * params->bpp + y * params->width*2 * params->bpp + 1] = out_bounds * 0 + (1-out_bounds) * (255 * white + (1-white) *src[(int)((int)(u) * params->bpp + (int)(v) * params->width*2 * params->bpp + 1)]);
    // dst[x * params->bpp + y * params->width*2 * params->bpp + 2] = out_bounds * 0 + (1-out_bounds) * (255 * white + (1-white) *src[(int)((int)(u) * params->bpp + (int)(v) * params->width*2 * params->bpp + 2)]);
    // dst[x * params->bpp + y * params->width*2 * params->bpp + 3] = 255;


	// int idx = blockIdx.x * blockDim.x + threadIdx.x;
	// if (idx*2 >= width) {
	// 	return;
	// }
	// int bytes_per_pixel = 3;
	// idx = 

	//for (int i = 0; i < height; ++i) {
		// int y0 = src[i*width*2+idx*4+0];
		// int cb = src[i*width*2+idx*4+1];
		// int y1 = src[i*width*2+idx*4+2];
		// int cr = src[i*width*2+idx*4+3];

		// dst[i*width*3+idx*6+0] = clamp(1.164f * (y0 - 16) + 1.596f * (cr - 128)                      , 0.0f, 255.0f);
		// dst[i*width*3+idx*6+1] = clamp(1.164f * (y0 - 16) - 0.813f * (cr - 128) - 0.391f * (cb - 128), 0.0f, 255.0f);
		// dst[i*width*3+idx*6+2] = clamp(1.164f * (y0 - 16)                       + 2.018f * (cb - 128), 0.0f, 255.0f);

		// dst[i*width*3+idx*6+3] = clamp(1.164f * (y1 - 16) + 1.596f * (cr - 128)                      , 0.0f, 255.0f);
		// dst[i*width*3+idx*6+4] = clamp(1.164f * (y1 - 16) - 0.813f * (cr - 128) - 0.391f * (cb - 128), 0.0f, 255.0f);
		// dst[i*width*3+idx*6+5] = clamp(1.164f * (y1 - 16)                       + 2.018f * (cb - 128), 0.0f, 255.0f);
	//}
}
#endif
__global__ void gpuConvertYUYVtoRGB_kernel(unsigned char *src, unsigned char *dst,
		unsigned int width, unsigned int height)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx*2 >= width) {
		return;
	}

	for (int i = 0; i < height; ++i) {
		int y0 = src[i*width*2+idx*4+0];
		int cb = src[i*width*2+idx*4+1];
		int y1 = src[i*width*2+idx*4+2];
		int cr = src[i*width*2+idx*4+3];

		dst[i*width*3+idx*6+0] = clamp(1.164f * (y0 - 16) + 1.596f * (cr - 128)                      , 0.0f, 255.0f);
		dst[i*width*3+idx*6+1] = clamp(1.164f * (y0 - 16) - 0.813f * (cr - 128) - 0.391f * (cb - 128), 0.0f, 255.0f);
		dst[i*width*3+idx*6+2] = clamp(1.164f * (y0 - 16)                       + 2.018f * (cb - 128), 0.0f, 255.0f);

		dst[i*width*3+idx*6+3] = clamp(1.164f * (y1 - 16) + 1.596f * (cr - 128)                      , 0.0f, 255.0f);
		dst[i*width*3+idx*6+4] = clamp(1.164f * (y1 - 16) - 0.813f * (cr - 128) - 0.391f * (cb - 128), 0.0f, 255.0f);
		dst[i*width*3+idx*6+5] = clamp(1.164f * (y1 - 16)                       + 2.018f * (cb - 128), 0.0f, 255.0f);
	}
}



void gpuConvertYUYVtoRGB(unsigned char *src, unsigned char *dst,
		unsigned int width, unsigned int height)
{
	unsigned char *d_src = NULL;
	unsigned char *d_dst = NULL;
	size_t planeSize = width * height * sizeof(unsigned char);

	unsigned int flags;
	bool srcIsMapped = (hipHostGetFlags(&flags, src) == hipSuccess) && (flags & hipHostMallocMapped);
	bool dstIsMapped = (hipHostGetFlags(&flags, dst) == hipSuccess) && (flags & hipHostMallocMapped);

	if (srcIsMapped) {
		d_src = src;
		hipStreamAttachMemAsync(NULL, src, 0, hipMemAttachGlobal);
	} else {
		hipMalloc(&d_src, planeSize * 2);
		hipMemcpy(d_src, src, planeSize * 2, hipMemcpyHostToDevice);
	}
	if (dstIsMapped) {
		d_dst = dst;
		hipStreamAttachMemAsync(NULL, dst, 0, hipMemAttachGlobal);
	} else {
		hipMalloc(&d_dst, planeSize * 3);
	}

	unsigned int blockSize = 1024;
	unsigned int numBlocks = (width / 2 + blockSize - 1) / blockSize;
	gpuConvertYUYVtoRGB_kernel<<<numBlocks, blockSize>>>(d_src, d_dst, width, height);
	hipMemcpy(dst, d_dst, planeSize * 3, hipMemcpyDeviceToHost);

	hipStreamAttachMemAsync(NULL, dst, 0, hipMemAttachHost);
	hipStreamSynchronize(NULL);

	if (!srcIsMapped) {
		hipMemcpy(dst, d_dst, planeSize * 3, hipMemcpyDeviceToHost);
		hipFree(d_src);
	}
	if (!dstIsMapped) {
		hipFree(d_dst);
	}
}
